#include "hip/hip_runtime.h"
#include "BitMap.h"
#include "stdio.h"
#if 1
#include<iostream>
using namespace std;
#endif
//========================================
//  Definition
//========================================.
#define BLOCKSIZE 32 	//1024 = 32*32 thread is a limit
#define MAX_DEPTH 0xFFFF
#define SIXTY_DEGREE 60
#define RGB_SCALE 100
#define HSV_SCALE 0.3
#define MAX_BIN 256

#define BOUND(x,min, max) ((x) > (max) ? (max): ((x) < (min)? (min): (x)))
#define MAXRGB(R,G,B) (R)>(G)?(R>B?R:B):((G)>(B)?(G):(B))
#define MINRGB(R,G,B) (R)<(G)?(R<B?R:B):((G<B)?G:B)
#define DEPTH_TO_INDEX(value) (value*255/MAX_DEPTH)
//========================================
//  CUDA Function
//========================================
__global__ void AdjustColorKernel(unsigned short* devPtr, int width, int height, int* RGBvalue)
{
	int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	int tidy = blockIdx.y * blockDim.y + threadIdx.y;
	int offset = tidx + tidy * width;
	
	if(tidx <= width && tidy <= height)
	{		
		devPtr[offset*4 + RED_PIXEL] = BOUND(devPtr[offset*4 + RED_PIXEL] + RGBvalue[RED_PIXEL]*(MAX_DEPTH/RGB_SCALE), 0, MAX_DEPTH);
		devPtr[offset*4 + GREEN_PIXEL] = BOUND(devPtr[offset*4 + GREEN_PIXEL] + RGBvalue[GREEN_PIXEL]*(MAX_DEPTH/RGB_SCALE), 0, MAX_DEPTH);
		devPtr[offset*4 + BLUE_PIXEL] = BOUND(devPtr[offset*4 + BLUE_PIXEL] + RGBvalue[BLUE_PIXEL]*(MAX_DEPTH/RGB_SCALE), 0, MAX_DEPTH);
	}
}

__device__ void ConvertHSVtoRGB(unsigned short* devPtr, int offset, float H, float S, float V)
{
	float R = 0, G = 0, B = 0;
	float h = (int)H/60;
	float f = H/60.0 - h;
	float p = V*(1-S);
	float q = V*(1-f*S);
	float t = V*(1-(1-f)*S);
	
	switch((int)h)
	{
		case 0:
			R = V;
			G = t;
			B = p;
			break;
		case 1:
			R = q;
			G = V;
			B = p;
			break;
		case 2:
			R = p;
			G = V;
			B = t;
			break;
		case 3:
			R = p;
			G = q;
			B = V;
			break;
		case 4:
			R = t;
			G = p;
			B = V;
			break;
		default:
			R = V;
			G = p;
			B = q;
			break;
	}	
	
	devPtr[offset*4] = R*MAX_DEPTH;
	devPtr[offset*4+1] = G*MAX_DEPTH;
	devPtr[offset*4+2] = B*MAX_DEPTH;
}

__global__ void ConvertRGBtoHSV(unsigned short* devPtr, int width, int height, int* HSVvalue)
{
	int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	int tidy = blockIdx.y * blockDim.y + threadIdx.y;
	int offset = tidx + tidy * width;
	
	if(tidx <= width && tidy <= height)
	{	
		float H = 0, S = 0, V = 0;
		float R = devPtr[offset*4];
		float G = devPtr[offset*4 + 1];
		float B = devPtr[offset*4 + 2];
		float max = MAXRGB(R,G,B);
		float min = MINRGB(R,G,B);
		float delta = max - min;
				
		//H
		if(min == max)
			H = 0;
		else if(max == R && G >= B)
			H=(SIXTY_DEGREE*(G-B)/delta);
		else if(max == R && G < B) 
			H=SIXTY_DEGREE*((G-B)/delta + 6);
		else if(max == G)
			H=SIXTY_DEGREE*((B-R)/delta + 2);
		else  //max == B
			H=SIXTY_DEGREE*((R-G)/delta +4);
			
		//Brightness
		V = max/MAX_DEPTH;
		
		//Saturation
		if(max == 0)
		{
			S = 0;
		}
		else
		{
			S = 1 - min/max;
		}
			
		//RED
		if(0 <= H && H < 15)
		{			
			H += 360;
			H = BOUND(H + HSVvalue[HUE_RED_TYPE], 330, 374);
						
			if(H >= 360)
				H -= 360;
			
			S = BOUND(S+HSVvalue[SAT_RED_TYPE]*0.01, 0, 1);
			V = BOUND(V+(float)HSVvalue[BRI_RED_TYPE]*0.01,0,1);
		}
		else if(315 <= H && H <= 360)
		{
			H = BOUND(H+ HSVvalue[HUE_RED_TYPE], 315, 360);
			if(H >= 360)
				H -= 360;
			
			S = BOUND(S+HSVvalue[SAT_RED_TYPE]*0.01, 0, 1);
			V = BOUND(V+(float)HSVvalue[BRI_RED_TYPE]*0.01,0,1);
		}
		else if(15 <= H && H < 45)
		{
			H = BOUND(H+HSVvalue[HUE_ORANGE_TYPE]*HSV_SCALE, 15, 44);
			S = BOUND(S+HSVvalue[SAT_ORANGE_TYPE]*0.01, 0, 1);
			V = BOUND(V+(float)HSVvalue[BRI_ORANGE_TYPE]*0.01,0,1);
		}
		else if(45 <= H && H < 75)
		{
			H = BOUND(H+HSVvalue[HUE_YELLOW_TYPE]*HSV_SCALE, 45, 74);
			S = BOUND(S+HSVvalue[SAT_YELLOW_TYPE]*0.01, 0, 1);
			V = BOUND(V+(float)HSVvalue[BRI_YELLOW_TYPE]*0.01,0,1);
		}
		else if(75 <= H && H < 165)
		{
			H = BOUND(H + HSVvalue[HUE_GREEN_TYPE]*HSV_SCALE, 75, 164);	
			S = BOUND(S+HSVvalue[SAT_GREEN_TYPE]*0.01, 0, 1);
			V = BOUND(V+(float)HSVvalue[BRI_GREEN_TYPE]*0.01,0,1);
		}
		else if(165 <= H && H < 225)
		{
			H = BOUND(H+HSVvalue[HUE_CYAN_TYPE]*HSV_SCALE, 165, 224);
			S = BOUND(S+HSVvalue[SAT_CYAN_TYPE]*0.01, 0, 1);
			V = BOUND(V+(float)HSVvalue[BRI_CYAN_TYPE]*0.01,0,1);
		}
		else if(225 <= H && H < 255)
		{
			H=BOUND(H+HSVvalue[HUE_BLUE_TYPE]*HSV_SCALE, 225, 254);	
			S = BOUND(S+HSVvalue[SAT_BLUE_TYPE]*0.01, 0, 1);
			V = BOUND(V+(float)HSVvalue[BRI_BLUE_TYPE]*0.01,0,1);
		}
		else if(255 <= H && H < 285)
		{
			H=BOUND(H+HSVvalue[HUE_VIOLET_TYPE]*HSV_SCALE, 255, 284);
			S = BOUND(S+HSVvalue[SAT_VIOLET_TYPE]*0.01, 0, 1);
			V = BOUND(V+(float)HSVvalue[BRI_VIOLET_TYPE]*0.01,0,1);
		}
		else if(285 <= H && H < 315)
		{	 
			H=BOUND(H+HSVvalue[HUE_MAGENTA_TYPE]*HSV_SCALE, 285, 314);
			S = BOUND(S+HSVvalue[SAT_MAGENTA_TYPE]*0.01, 0, 1);
			V = BOUND(V+(float)HSVvalue[BRI_MAGENTA_TYPE]*0.01,0,1);
		}
						
		ConvertHSVtoRGB(devPtr, offset, H, S, V);
	}
}

__global__ void getHistogram(unsigned short* devPtr, int width, int height, int* devHistogram)
{
	//tidx, tidy = 0~31
	int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	int tidy = blockIdx.y * blockDim.y + threadIdx.y;

	//thread in the "same blocks", which is different from tidx, tidy 
	//,using same share memory
	//0~1023
	int t = threadIdx.y * blockDim.x + threadIdx.x;
	//nt: total thread per block: 32*32=1024
	int nt = blockDim.x * blockDim.y;

	int offset = tidx + tidy * width;
	unsigned short RIndex, GIndex, BIndex;

	__shared__ unsigned int Rtemp[MAX_BIN];
	//__shared__ unsigned int Gtemp[MAX_BIN];
	//__shared__ unsigned int Btemp[MAX_BIN];

	//same block using same share memory
	//even each block only 1024 threads, array size > 1024(block size)
	//each thread must initialize more than one location
	if(MAX_BIN > BLOCKSIZE)
	{
		// t represent for all thread in the same block
		//+nt for next block
		for(int i = t; i < MAX_BIN; i+=nt)
		{
			if(tidx < MAX_BIN)
				Rtemp[i] = 0;
			//Gtemp[i] = 0;
			//Btemp[i] = 0;
		}
	}
	else
	{
		//use the first thread of each block to init
		if(t < MAX_BIN)
			Rtemp[t] = 0;
	}

	__syncthreads();

	if(tidx < width && tidy < height)
	{
		RIndex = DEPTH_TO_INDEX(devPtr[offset*4 + RED_PIXEL]);
		//GIndex = devPtr[offset*4 + GREEN_PIXEL];
		//BIndex = devPtr[offset*4 + BLUE_PIXEL];
		//printf("RIndex:%d, devPtr[offset*4 + RED_PIXEL]:%d\n", RIndex, devPtr[offset*4 + RED_PIXEL]);

		atomicAdd(&Rtemp[RIndex], 1);
		//atomicAdd(&Gtemp[GIndex], 1);
		//atomicAdd(&Btemp[BIndex], 1);
	}
	__syncthreads();

	if(MAX_BIN > BLOCKSIZE)
	{
		// t represent for all thread in the same block
		//+nt for next block
		for(int i = t; i < MAX_BIN; i+=nt)
		{
			if(i < MAX_BIN)
				atomicAdd(&devHistogram[i], Rtemp[i]);
			//Gtemp[i] = 0;
			//Btemp[i] = 0;
		}
	}
	else
	{
		//use the first thread of each block to init
		if(t < MAX_BIN)
			atomicAdd(&devHistogram[t], Rtemp[t]);
	}
}

//========================================
//  Function
//========================================
Histogram::Histogram(int sum)
{
	RHist = new float[sum]{0};
	GHist = new float[sum];
	BHist = new float[sum];
	YHist = new float[sum];
}

Histogram::~Histogram()
{
	if(RHist)
		delete [] RHist;
	if(GHist)
		delete [] GHist;
	if(BHist)
		delete [] BHist;
	if(YHist)	
		delete [] YHist;
}

BitMap::BitMap(unsigned short* _data, int _width, int _height, int _channel, int _depth)
	:data(_data),
	width(_width),
	height(_height),
	channel(_channel),
	depth(_depth),
	info(_width*_height)
{
	//binSize = 1<<depth;
	binSize = MAX_BIN;
	size = width*height*channel*sizeof(unsigned short);	

	proImage = new unsigned short[size];
	memcpy(proImage , data, size);
}

void BitMap::refresh()
{
	memcpy(proImage, data, size);
}

void BitMap::updateHistogram()
{
	int bx = (width + BLOCKSIZE - 1)/BLOCKSIZE;
	int by = (height + BLOCKSIZE - 1)/BLOCKSIZE;
	dim3 gridSize(bx, by);
	dim3 blockSize(BLOCKSIZE, BLOCKSIZE);
	unsigned short* devPtr;
	//Histogram* devHistogram;
	int* devRHist;

	//histogram malloc
	//hipMalloc((void**)&devHistogram.RHist, sizeof(float)*width*height);
	//hipMemset( devHistogram.RHist, 0x0, sizeof(float)*width*height);
	hipMalloc((void**)&devRHist, sizeof(int)*MAX_BIN);
	hipMemset(devRHist, 0x0, sizeof(int)*MAX_BIN);

	//image malloc
	hipMalloc( (void**)&devPtr, size);
	hipMemcpy( devPtr, proImage, size, hipMemcpyHostToDevice);

	//prefer shared memory larger than L1 cache
	//hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	//getHistogram<<<gridSize, blockSize>>>(devPtr, width, height, devHistogram.RHist);
	getHistogram<<<gridSize, blockSize>>>(devPtr, width, height, devRHist);

	//copy histogram
	//hipMemcpy(info.RHist, devHistogram.RHist, sizeof(float)*width*height , hipMemcpyDeviceToHost );

	int data[MAX_BIN] = {0};
	hipMemcpy(data, devRHist, sizeof(int)*MAX_BIN , hipMemcpyDeviceToHost );
	
	//hipMemcpy(info.RHist, devRHist, sizeof(float)*width*height , hipMemcpyDeviceToHost );
	for(int i = 0; i < MAX_BIN; i++)
		cout<<"i:"<<i<<" rhis:"<<data[i]<<endl;

	hipFree(devRHist);
	hipFree(devPtr);	
}

void Processor::process(BitMap* bitmap, COLOR_ITEMS uiSlider)
{
	if(memcmp(&slider, &uiSlider, sizeof(COLOR_ITEMS)) == 0)
	{
		return;
	}
	
	//to copy the original image to new one for processing
	bitmap->refresh();
	
	//RGB 
	AdjustColor(bitmap, uiSlider.RGBvalue);

	//HSY
	AdjustHSV(bitmap,  uiSlider.HSVvalue);
	
	bitmap->updateHistogram();

	memcpy(&slider, &uiSlider, sizeof(COLOR_ITEMS));
}

void Processor::AdjustColor(BitMap* bitmap, int* RGBvalue)
{
	unsigned short* devPtr;
	int* devRGBvalue;
	int height = bitmap->get_height();
	int width = bitmap->get_width();
	int size = bitmap->get_size();
		
	hipMalloc((void**)&devRGBvalue, sizeof(int)*MAX_PIXEL);
	hipMemcpy( devRGBvalue, RGBvalue, sizeof(int)*MAX_PIXEL ,hipMemcpyHostToDevice );
	
	hipMalloc( (void**)&devPtr, size );
	hipMemcpy( devPtr, bitmap->get_Image(), size,hipMemcpyHostToDevice );

	int bx = (width + BLOCKSIZE - 1)/BLOCKSIZE;
	int by = (height + BLOCKSIZE - 1)/BLOCKSIZE;
	dim3 gridSize(bx, by);
	dim3 blockSize(BLOCKSIZE, BLOCKSIZE);
		
	AdjustColorKernel<<<gridSize, blockSize>>>(devPtr, width, height, devRGBvalue);
	
	hipMemcpy( bitmap->get_Image(), devPtr, size,hipMemcpyDeviceToHost );
	hipFree(devRGBvalue);
	hipFree(devPtr);
}

void CPUHSVtoRGB(unsigned short* devPtr, float H, float S, float V)
{
	float R, G, B = 0;
	float h = (int)H/60;
	float f = H/60.0 - h;
	float p = V*(1-S);
	float q = V*(1-f*S);
	float t = V*(1-(1-f)*S);
	
	switch((int)h)
	{
		case 0:
			R = V;
			G = t;
			B = p;
			break;
		case 1:
			R = q;
			G = V;
			B = p;
			break;
		case 2:
			R = p;
			G = V;
			B = t;
			break;
		case 3:
			R = p;
			G = q;
			B = V;
			break;
		case 4:
			R = t;
			G = p;
			B = V;
			break;
		default:
			R = V;
			G = p;
			B = q;
			break;
	}

	devPtr[0] = R*MAX_DEPTH;
	devPtr[1] = G*MAX_DEPTH;
	devPtr[2] = B*MAX_DEPTH;	
	//cout<<"R:"<<R<<" G:"<<G<<" B:"<<B<<endl;
}

void CPUConvertRGBtoHSV(unsigned short* devPtr, int width, int height, int* HSVvalue)
{
		float H = 0, S = 0, V = 0;

		for(int y = 0; y< height; y++)
		{
			for(int x= 0; x< width; x++)
			{
				int offset = y*width+x;
				float R = devPtr[offset*4];
				float G = devPtr[offset*4 + 1];
				float B = devPtr[offset*4 + 2];
				float max = MAXRGB(R,G,B);
				float min = MINRGB(R,G,B);
				float delta = max - min;
				
				//H
				if(min == max)
					H = 0;
				else if(max == R && G >= B)
					H=(SIXTY_DEGREE*(G-B)/delta);
				else if(max == R && G < B) 
					H=SIXTY_DEGREE*((G-B)/delta + 6);
				else if(max == G)
					H=SIXTY_DEGREE*((B-R)/delta + 2);
				else  //max == B
					H=SIXTY_DEGREE*((R-G)/delta +4);				
					
				//Brightness
				V = max/MAX_DEPTH;
				
				//Saturation
				if(max == 0)
				{
					S = 0;
				}
				else
				{
					S = 1 - min/max;
				}
					
				//RED
				if(0 <= H && H < 15)
				{			
					H += 360;
					H = BOUND(H + HSVvalue[HUE_RED_TYPE], 330, 374);
								
					if(H >= 360)
						H -= 360;
					
					S = BOUND(S+HSVvalue[SAT_RED_TYPE]*0.01, 0, 1);
					V = BOUND(V+(float)HSVvalue[BRI_RED_TYPE]*0.01,0,1);
				}
				else if(315 <= H && H <= 360)
				{
					H = BOUND(H+ HSVvalue[HUE_RED_TYPE], 315, 360);
					if(H >= 360)
						H -= 360;
					
					S = BOUND(S+HSVvalue[SAT_RED_TYPE]*0.01, 0, 1);
					V = BOUND(V+(float)HSVvalue[BRI_RED_TYPE]*0.01,0,1);
				}
				else if(15 <= H && H < 45)
				{
					H = BOUND(H+HSVvalue[HUE_ORANGE_TYPE]*HSV_SCALE, 15, 44);
					S = BOUND(S+HSVvalue[SAT_ORANGE_TYPE]*0.01, 0, 1);
					V = BOUND(V+(float)HSVvalue[BRI_ORANGE_TYPE]*0.01,0,1);
				}
				else if(45 <= H && H < 75)
				{
					H = BOUND(H+HSVvalue[HUE_YELLOW_TYPE]*HSV_SCALE, 45, 74);
					S = BOUND(S+HSVvalue[SAT_YELLOW_TYPE]*0.01, 0, 1);
					V = BOUND(V+(float)HSVvalue[BRI_YELLOW_TYPE]*0.01,0,1);
				}
				else if(75 <= H && H < 165)
				{
					H = BOUND(H + HSVvalue[HUE_GREEN_TYPE]*HSV_SCALE, 75, 164);	
					S = BOUND(S+HSVvalue[SAT_GREEN_TYPE]*0.01, 0, 1);
					V = BOUND(V+(float)HSVvalue[BRI_GREEN_TYPE]*0.01,0,1);
				}
				else if(165 <= H && H < 225)
				{
					H = BOUND(H+HSVvalue[HUE_CYAN_TYPE]*HSV_SCALE, 165, 224);
					S = BOUND(S+HSVvalue[SAT_CYAN_TYPE]*0.01, 0, 1);
					V = BOUND(V+(float)HSVvalue[BRI_CYAN_TYPE]*0.01,0,1);
				}
				else if(225 <= H && H < 255)
				{
					H=BOUND(H+HSVvalue[HUE_BLUE_TYPE]*HSV_SCALE, 225, 254);	
					S = BOUND(S+HSVvalue[SAT_BLUE_TYPE]*0.01, 0, 1);
					V = BOUND(V+(float)HSVvalue[BRI_BLUE_TYPE]*0.01,0,1);
				}
				else if(255 <= H && H < 285)
				{
					H=BOUND(H+HSVvalue[HUE_VIOLET_TYPE]*HSV_SCALE, 255, 284);
					S = BOUND(S+HSVvalue[SAT_VIOLET_TYPE]*0.01, 0, 1);
					V = BOUND(V+(float)HSVvalue[BRI_VIOLET_TYPE]*0.01,0,1);
				}
				else if(285 <= H && H < 315)
				{	 
					H=BOUND(H+HSVvalue[HUE_MAGENTA_TYPE]*HSV_SCALE, 285, 314);
					S = BOUND(S+HSVvalue[SAT_MAGENTA_TYPE]*0.01, 0, 1);
					V = BOUND(V+(float)HSVvalue[BRI_MAGENTA_TYPE]*0.01,0,1);
				}
				
				CPUHSVtoRGB(&devPtr[offset*4], H, S, V);
			}
		}
}


void Processor::AdjustHSV(BitMap* bitmap, int* HSVvalue)
{
	unsigned short* devPtr;
	int* devHSVvalue;
	int height = bitmap->get_height();
	int width = bitmap->get_width();
	int size = bitmap->get_size();
	
	//CPUConvertRGBtoHSV(bitmap->get_Image(), width, height, HSVvalue);
	
	hipMalloc((void**)&devHSVvalue, sizeof(int)*HSV_MAX_TYPE);
	hipMemcpy( devHSVvalue, HSVvalue, sizeof(int)*HSV_MAX_TYPE, hipMemcpyHostToDevice );
	
	hipMalloc( (void**)&devPtr, size );
	hipMemcpy( devPtr, bitmap->get_Image(), size,hipMemcpyHostToDevice );

	int bx = (width + BLOCKSIZE - 1)/BLOCKSIZE;
	int by = (height + BLOCKSIZE - 1)/BLOCKSIZE;
	dim3 gridSize(bx, by);
	dim3 blockSize(BLOCKSIZE, BLOCKSIZE);
	
	ConvertRGBtoHSV<<<gridSize, blockSize>>>(devPtr, width, height, devHSVvalue);
	
	hipMemcpy(bitmap->get_Image(), devPtr, size,hipMemcpyDeviceToHost );

	hipFree(devHSVvalue);
	hipFree(devPtr);	
}
